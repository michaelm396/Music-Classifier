#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"

#include "knn_cuda_kernel.h"

// Constants used by the program
#define BLOCK_DIM                      16


/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  *
  * @param A     pointer on the matrix A
  * @param wA    width of the matrix A = number of points in A
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */

__global__ void EuclidianDistances( float* A, int n,
    float* B, int m, int dim, float* C )
{
        // SIZE is equal to 128
	__shared__ float accumResult[dim];
	float sA;
	float sB;

        // MAPPING
	int bx = blockIdx.x;  // n
	int by = blockIdx.y;  // m
	int ty = threadIdx.y; // dim
	int tx = threadIdx.x; // 1


	sA = A [bx * n + ty];
	sB = B [by * m + ty];
	__syncthreads();


	accumResult[ty] = (sA - sB)*(sA - sB);
	__syncthreads();


	// Parallel tree-reduction
	for (int i = dim/2 ; i > 0 ; i >>= 1)
 {
  if (ty < pas)
			accumResult[ty]	+= accumResult [i + ty];
	__syncthreads();
 }
 if ((threadIdx.y == 0))
		C [bx * m + by] = accumResult[ty];
	__syncthreads();
}



/**
  * Gathers k-th smallest distances for each column of the distance matrix in the top.
  *
  * @param dist        distance matrix
  * @param ind         index matrix
  * @param width       width of the distance matrix and of the index matrix
  * @param height      height of the distance matrix and of the index matrix
  * @param k           number of neighbors to consider
  */
__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k){

  // Variables
  int l, i, j;
  float *p_dist;
  long  *p_ind;
  float curr_dist, max_dist;
  long  curr_row,  max_row;
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (xIndex<width){
    // Pointer shift, initialization, and max value
    p_dist   = dist + xIndex;
    p_ind    = ind  + xIndex;
    max_dist = p_dist[0];
    p_ind[0] = 1;

    // Part 1 : sort kth firt elementZ
    for (l=1; l<k; l++){
      curr_row  = l * width;
      curr_dist = p_dist[curr_row];
      if (curr_dist<max_dist){
        i=l-1;
        for (int a=0; a<l-1; a++){
          if (p_dist[a*width]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=l; j>i; j--){
          p_dist[j*width] = p_dist[(j-1)*width];
          p_ind[j*width]   = p_ind[(j-1)*width];
        }
        p_dist[i*width] = curr_dist;
        p_ind[i*width]   = l+1;
      } else {
        p_ind[l*width] = l+1;
      }
      max_dist = p_dist[curr_row];
    }

    // Part 2 : insert element in the k-th first lines
    max_row = (k-1)*width;
    for (l=k; l<height; l++){
      curr_dist = p_dist[l*width];
      if (curr_dist<max_dist){
        i=k-1;
        for (int a=0; a<k-1; a++){
          if (p_dist[a*width]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=k-1; j>i; j--){
          p_dist[j*width] = p_dist[(j-1)*width];
          p_ind[j*width]   = p_ind[(j-1)*width];
        }
        p_dist[i*width] = curr_dist;
        p_ind[i*width]   = l+1;
        max_dist             = p_dist[max_row];
      }
    }
  }
}




/**
  * K nearest neighbor algorithm
  * - Initialize CUDA
  * - Allocate device memory
  * - Copy point sets (reference and query points) from host to device memory
  * - Compute the distances + indexes to the k nearest neighbors for each query point
  * - Copy distances from device to host memory
  *
  * @param ref_host      reference points ; pointer to linear matrix
  * @param ref_nb        number of reference points ; width of the matrix
  * @param query_host    query points ; pointer to linear matrix
  * @param query_nb      number of query points ; width of the matrix
  * @param dim           dimension of points ; height of the matrices
  * @param k             number of neighbor to consider
  * @param dist_host     distances to k nearest neighbors ; pointer to linear matrix
  * @param dist_host     indexes of the k nearest neighbors ; pointer to linear matrix
  *
  */
void knn_device(float* ref_dev, int ref_nb, float* query_dev, int query_nb,
    int dim, int k, float* dist_dev, long* ind_dev, hipStream_t stream){

  // Grids ans threads
  dim3 g_16x16(query_nb, ref_nb, 1);
  dim3 t_k_16x16(dim,1,1);
  //
  dim3 g_256x1(query_nb/256, 1, 1);
  dim3 t_256x1(256, 1, 1);
  if (query_nb%256 != 0) g_256x1.x += 1;

   // Kernel 1: Compute all the distances
   EuclidianDistances<<<g_16x16, t_k_16x16, 0, stream>>>(ref_dev, ref_nb,
      query_dev, query_nb, dim, dist_dev);
  // Kernel 2: Sort each column
  cuInsertionSort<<<g_256x1, t_256x1, 0, stream>>>(dist_dev, ind_dev,
      query_nb, ref_nb, k);

}
